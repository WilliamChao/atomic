#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA

#include <GL/glew.h>
#include <hip/hip_runtime.h>
#include <cudaGL.h>
#include <cuda_gl_interop.h>
#include <hip/hip_math_constants.h>
#include <cutil.h>
#include <cutil_math.h>
#include <cutil_inline_runtime.h>

#include <thrust/device_ptr.h>
#include <thrust/sort.h>

#include "SPH.cuh"
#include "SPH_internal.cuh"
#include "../Graphics/ResourceID.h"

__constant__ sphParam d_params;

#include "SPHRigid.cuh"
#include "SPHForce.cuh"
#include "SPHFluid.cuh"


FluidDataSet *h_fluid = NULL;
RigidDataSet *h_rigid = NULL;
ForceDataSet *h_forces = NULL;

sphStates h_states;
thrust::host_vector<sphFluidParticle>       h_fluid_append;
thrust::host_vector<sphFluidMessage>        h_fluid_message;

DeviceBufferObject h_fluid_gl;


struct _ClearParticles
{
    DeviceFluidDataSet dfd;
    _ClearParticles(const DeviceFluidDataSet& v) : dfd(v) {}

    __device__ void operator()(int i)
    {
        const float spacing = 0.009f;
        uint w = 128;
        dfd.particles[i].position = make_float4(
            spacing*(i%w) - (spacing*w*0.5f),
            spacing*((i/w)%w) + 0.6,
            spacing*(i/(w*w))+0.05f,
            0.0f);
        dfd.particles[i].velocity = make_float4(0.0f);
        dfd.particles[i].density = 0.0f;
        dfd.particles[i].energy = 100.0f;
        dfd.message[i].to = 0;
    }
};

void SPHInitialize()
{
    h_fluid = new FluidDataSet();
    h_rigid = new RigidDataSet();
    h_forces = new ForceDataSet();

    {
        sphParam sph_params;
        sph_params.smooth_len           = 0.02f;
        sph_params.pressure_stiffness   = 200.0f;
        sph_params.rest_density         = 1000.0f;
        sph_params.particle_mass        = 0.001f;
        sph_params.viscosity            = 0.1f;
        sph_params.density_coef         = sph_params.particle_mass * 315.0f / (64.0f * HIP_PI_F * pow(sph_params.smooth_len, 9));
        sph_params.grad_pressure_coef   = sph_params.particle_mass * -45.0f / (HIP_PI_F * pow(sph_params.smooth_len, 6));
        sph_params.lap_viscosity_coef   = sph_params.particle_mass * sph_params.viscosity * 45.0f / (HIP_PI_F * pow(sph_params.smooth_len, 6));
        sph_params.wall_stiffness       = 3000.0f;
        CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL("d_params"), &sph_params, sizeof(sph_params)) );

        sphGridParam grid_params;
        const float grid_len = 5.12f;
        grid_params.grid_dim = make_float4(grid_len, grid_len, sph_params.smooth_len*SPH_FLUID_GRID_DIV_Z, 0.0f);
        grid_params.grid_dim_rcp = make_float4(1.0f) / (grid_params.grid_dim / make_float4(SPH_FLUID_GRID_DIV_X, SPH_FLUID_GRID_DIV_Y, SPH_FLUID_GRID_DIV_Z, 1.0));
        grid_params.grid_pos = make_float4(-grid_len/2.0f, -grid_len/2.0f, 0.0f, 0.0f);
        h_fluid->params[0] = grid_params;

        h_states.fluid_num_particles = 30000;
        h_states.fluid_alive_any = 0;
        h_fluid->states[0] = h_states;
    }

    h_fluid->resizeParticles(h_states.fluid_num_particles);
    thrust::for_each( thrust::make_counting_iterator(0), thrust::make_counting_iterator(h_states.fluid_num_particles), _ClearParticles(h_fluid->getDeviceData()) );
}

void SPHFinalize()
{
    delete h_forces;h_forces=NULL;
    delete h_rigid; h_rigid=NULL;
    delete h_fluid; h_fluid=NULL;
}


void SPHInitializeGLBuffers(int vbo_fluid)
{
    h_fluid_gl.registerBuffer(vbo_fluid, cudaGraphicsMapFlagsWriteDiscard);
}

void SPHFinalizeGLBuffers()
{
    h_fluid_gl.unregisterBuffer();
}



void SPHUpdateRigids(
    const thrust::host_vector<sphRigidPlane> &planes,
    const thrust::host_vector<sphRigidSphere> &spheres,
    const thrust::host_vector<sphRigidBox> &boxes
    )
{
    h_rigid->planes = planes;
    h_rigid->spheres = spheres;
    h_rigid->boxes = boxes;
}


void SPHUpdateForce(const thrust::host_vector<sphForcePointGravity> &pgravity)
{
    h_forces->point_gravity = pgravity;
}

void SPHAddFluid(const thrust::host_vector<sphFluidParticle> &particles)
{
    if(particles.empty()) { return; }

    uint current = h_fluid->particles.size();
    h_fluid->resizeParticles(current + particles.size());
    thrust::copy(particles.begin(), particles.end(), h_fluid->particles.begin()+current);
    h_states.fluid_num_particles = h_fluid->particles.size();
    h_fluid->states[0] = h_states;
}

void SPHUpdateFluid()
{
    DeviceFluidDataSet dfd = h_fluid->getDeviceData();
    DeviceRigidDataSet drd = h_rigid->getDeviceData();
    DeviceForceDataSet dgd = h_forces->getDeviceData();
    int num_particles = (int)h_fluid->particles.size();

    thrust::for_each( thrust::make_counting_iterator(0), thrust::make_counting_iterator(num_particles), _FluidUpdateHash(dfd) );
    thrust::sort_by_key(h_fluid->hashes.begin(), h_fluid->hashes.end(), h_fluid->particles.begin());
    thrust::for_each( thrust::make_counting_iterator(0), thrust::make_counting_iterator(SPH_FLUID_GRID_DIV_3), _FluidClearGrid(dfd));
    thrust::for_each( thrust::make_counting_iterator(0), thrust::make_counting_iterator(num_particles), _FluidUpdateGrid(dfd));

    thrust::for_each( thrust::make_counting_iterator(0), thrust::make_counting_iterator(num_particles), _FluidComputeDensity(dfd));
    thrust::for_each( thrust::make_counting_iterator(0), thrust::make_counting_iterator(num_particles), _FluidComputeForce(dfd));
    thrust::for_each( thrust::make_counting_iterator(0), thrust::make_counting_iterator(num_particles), _FluidIntegrate(dfd, drd, dgd));
    h_fluid_message = h_fluid->message;

    sphStates stat = h_fluid->states[0];
    h_states.fluid_num_particles = stat.fluid_alive_any==0 ? 0 : stat.fluid_num_particles;
    h_states.fluid_alive_any = 0;
    h_fluid->states[0] = h_states;
    h_fluid->resizeParticles(stat.fluid_num_particles);
}


void SPHCopyToGL()
{
    sphFluidParticle *gl_fluid = (sphFluidParticle*)h_fluid_gl.mapBuffer();
    thrust::copy(h_fluid->particles.begin(), h_fluid->particles.end(), thrust::device_ptr<sphFluidParticle>(gl_fluid));
    h_fluid_gl.unmapBuffer();
}


const sphStates& SPHGetStates()
{
    return h_states;
}

const thrust::host_vector<sphFluidMessage>& SPHGetFluidMessage()
{
    return h_fluid_message;
}
