#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA

#include <GL/glew.h>
#include <hip/hip_runtime.h>
#include <cudaGL.h>
#include <cuda_gl_interop.h>
#include <hip/hip_math_constants.h>
#include <cutil.h>
#include <cutil_math.h>
#include <cutil_inline_runtime.h>

#include <thrust/device_ptr.h>
#include <thrust/sort.h>

#include "SPH.cuh"
#include "SPH_internal.cuh"
#include "../Graphics/ResourceID.h"



__constant__ sphParam d_params;

struct DeviceFluidDataSet
{
    sphGridParam            *params;
    sphFluidParticle        *particles;
    sphFluidParticleForce   *forces;
    sphHash                 *hashes;
    sphGridData             *grid;
    sphStates               *states;

    __device__ int3 GridCalculateCell(float4 pos)
    {
        float4 c = (pos-params->grid_pos)*params->grid_dim_rcp;
        int3 uc = make_int3(c.x, c.y, c.z);
        return clamp(uc, make_int3(0), make_int3(SPH_FLUID_GRID_DIV_X-1, SPH_FLUID_GRID_DIV_Y-1, SPH_FLUID_GRID_DIV_Z-1));
    }

    __device__ uint GridCalculateHash(float4 pos)
    {
        return GridConstuctKey( GridCalculateCell(pos) );
    }

    __device__ uint GridConstuctKey(int3 v)
    {
        return v.x | (v.y<<SPH_FLUID_GRID_DIV_SHIFT_X) | (v.z<<(SPH_FLUID_GRID_DIV_SHIFT_X+SPH_FLUID_GRID_DIV_SHIFT_Y));
    }


    __device__ float CalculatePressure(float density)
    {
        // Implements this equation:
        // Pressure = B * ((rho / rho_0)^y  - 1)
        return d_params.pressure_stiffness * max(pow(density / d_params.rest_density, 3) - 1.0f, 0.0f);
    }

    __device__ float4 CalculateGradPressure(float r, float P_pressure, float N_pressure, float N_density, float4 diff)
    {
        const float h = d_params.smooth_len;
        float avg_pressure = 0.5f * (N_pressure + P_pressure);
        // Implements this equation:
        // W_spkiey(r, h) = 15 / (pi * h^6) * (h - r)^3
        // GRAD( W_spikey(r, h) ) = -45 / (pi * h^6) * (h - r)^2
        // g_fGradPressureCoef = fParticleMass * -45.0f / (PI * fSmoothlen^6)
        return (d_params.grad_pressure_coef * avg_pressure / N_density * (h - r) * (h - r) / r) * diff;
    }

    __device__ float4 CalculateLapVelocity(float r, float4 P_velocity, float4 N_velocity, float N_density)
    {
        const float h = d_params.smooth_len;
        float4 vel_diff = N_velocity - P_velocity;
        // Implements this equation:
        // W_viscosity(r, h) = 15 / (2 * pi * h^3) * (-r^3 / (2 * h^3) + r^2 / h^2 + h / (2 * r) - 1)
        // LAPLACIAN( W_viscosity(r, h) ) = 45 / (pi * h^6) * (h - r)
        // g_fLapViscosityCoef = fParticleMass * fViscosity * 45.0f / (PI * fSmoothlen^6)
        return  (d_params.lap_viscosity_coef / N_density * (h - r)) * vel_diff;
    }

    __device__ float CalculateDensity(float r_sq)
    {
        const float h_sq = d_params.smooth_len * d_params.smooth_len;
        // Implements this equation:
        // W_poly6(r, h) = 315 / (64 * pi * h^9) * (h^2 - r^2)^3
        // g_fDensityCoef = fParticleMass * 315.0f / (64.0f * PI * fSmoothlen^9)
        return d_params.density_coef * (h_sq - r_sq) * (h_sq - r_sq) * (h_sq - r_sq);
    }



    __device__ void updateHash(int i)
    {
        hashes[i] = GridCalculateHash(particles[i].position);
    }

    __device__ void clearGrid(int i)
    {
        grid[i].x = grid[i].y = 0;
    }

    __device__ void updateGrid(int i)
    {
        const uint G_ID = i;
        uint G_ID_PREV = (G_ID == 0)? SPH_MAX_FLUID_PARTICLES : G_ID; G_ID_PREV--;
        uint G_ID_NEXT = G_ID + 1; if (G_ID_NEXT == SPH_MAX_FLUID_PARTICLES) { G_ID_NEXT = 0; }
    
        uint cell = hashes[G_ID];
        uint cell_prev = hashes[G_ID_PREV];
        uint cell_next = hashes[G_ID_NEXT];
        if (cell != cell_prev)
        {
            // I'm the start of a cell
            grid[cell].x = G_ID;
        }
        if (cell != cell_next)
        {
            // I'm the end of a cell
            grid[cell].y = G_ID + 1;
        }
    }

    __device__ void computeDensity(int i)
    {
        const uint P_ID = i;
        const float h_sq = d_params.smooth_len * d_params.smooth_len;
        float4 P_position = particles[P_ID].position;

        float density = 0.0f;

        int3 G_XYZ = GridCalculateCell( P_position );
        for(int Z = max(G_XYZ.z - 1, 0) ; Z <= min(G_XYZ.z + 1, SPH_FLUID_GRID_DIV_Z-1) ; Z++)
        {
            for(int Y = max(G_XYZ.y - 1, 0) ; Y <= min(G_XYZ.y + 1, SPH_FLUID_GRID_DIV_Y-1) ; Y++)
            {
                for(int X = max(G_XYZ.x - 1, 0) ; X <= min(G_XYZ.x + 1, SPH_FLUID_GRID_DIV_X-1) ; X++)
                {
                    sphHash G_CELL = GridConstuctKey(make_int3(X, Y, Z));
                    sphGridData G_START_END = grid[G_CELL];
                    for(uint N_ID = G_START_END.x ; N_ID < G_START_END.y ; N_ID++)
                    {
                        float4 N_position = particles[N_ID].position;
                
                        float4 diff = N_position - P_position;
                        float r_sq = dot(diff, diff);
                        if(r_sq < h_sq)
                        {
                            density += CalculateDensity(r_sq);
                        }
                    }
                }
            }
        }

        forces[P_ID].density = density;
    }

    __device__ void computeForce(int i)
    {
        const uint P_ID = i;
    
        float4 P_position = particles[P_ID].position;
        float4 P_velocity = particles[P_ID].velocity;
        float P_density = forces[P_ID].density;
        float P_pressure = CalculatePressure(P_density);
    
        const float h_sq = d_params.smooth_len * d_params.smooth_len;
    
        float4 acceleration = make_float4(0);

        // Calculate the acceleration based on all neighbors
        int3 G_XYZ = GridCalculateCell( P_position );
        for(int Z = max(G_XYZ.z - 1, 0) ; Z <= min(G_XYZ.z + 1, SPH_FLUID_GRID_DIV_Z-1) ; Z++)
        {
            for(int Y = max(G_XYZ.y - 1, 0) ; Y <= min(G_XYZ.y + 1, SPH_FLUID_GRID_DIV_Y-1) ; Y++)
            {
                for(int X = max(G_XYZ.x - 1, 0) ; X <= min(G_XYZ.x + 1, SPH_FLUID_GRID_DIV_X-1) ; X++)
                {
                    sphHash G_CELL = GridConstuctKey(make_int3(X, Y, Z));
                    sphGridData G_START_END = grid[G_CELL];
                    for(uint N_ID = G_START_END.x ; N_ID < G_START_END.y ; N_ID++)
                    {
                        float4 N_position = particles[N_ID].position;

                        float4 diff = N_position - P_position;
                        float r_sq = dot(diff, diff);
                        if(r_sq < h_sq && P_ID != N_ID)
                        {
                            float4 N_velocity = particles[N_ID].velocity;
                            float N_density = forces[N_ID].density;
                            float N_pressure = CalculatePressure(N_density);
                            float r = sqrt(r_sq);

                            // Pressure Term
                            acceleration += CalculateGradPressure(r, P_pressure, N_pressure, N_density, diff);

                            // Viscosity Term
                            acceleration += CalculateLapVelocity(r, P_velocity, N_velocity, N_density);
                        }
                    }
                }
            }
        }

        forces[P_ID].acceleration = acceleration / P_density;
    }

    __device__ void countAlives(int i)
    {
        const uint G_ID = i;
        uint G_ID_NEXT = G_ID + 1; if (G_ID_NEXT == SPH_MAX_FLUID_PARTICLES) { G_ID_NEXT--; }

        if (hashes[G_ID] != hashes[G_ID_NEXT]) {
            states[0].num_fluid_particles = G_ID + 1;
        }
    }
};

struct DeviceRigidDataSet
{
    sphGridParam            *params;
    sphStates               *states;
    sphRigidClass           *classinfo;
    sphRigidInstance        *instances;
    sphRigidParticle        *particles;
    sphRigidUpdateInfo      *updateinfo;
    sphHash                 *hashes;
    sphGridData             *grid;

    __device__ int3 GridCalculateCell(float4 pos)
    {
        float4 c = (pos-params->grid_pos)*params->grid_dim_rcp;
        int3 uc = make_int3(c.x, c.y, c.z);
        return clamp(uc, make_int3(0), make_int3(SPH_RIGID_GRID_DIV_X-1, SPH_RIGID_GRID_DIV_Y-1, SPH_RIGID_GRID_DIV_Z-1));
    }

    __device__ uint GridCalculateHash(float4 pos)
    {
        return GridConstuctKey( GridCalculateCell(pos) );
    }

    __device__ uint GridConstuctKey(int3 v)
    {
        return v.x | (v.y<<SPH_RIGID_GRID_DIV_SHIFT_X) | (v.z<<(SPH_RIGID_GRID_DIV_SHIFT_X+SPH_RIGID_GRID_DIV_SHIFT_Y));
    }

    __device__ void updateRigids(int i)
    {
        sphRigidUpdateInfo  &rui    = updateinfo[i];
        sphRigidClass       &rc     = classinfo[rui.classid];
        sphRigidInstance    &rin    = instances[rui.cindex];
        sphRigidParticle    &rp     = rc.particles[rui.pindex];
        particles[i].owner_handle   = rui.owner_handle;
        particles[i].position       = vector_cast<float4&>(rin.transform * vector_cast<vec4>(rp.position));
        particles[i].normal         = vector_cast<float4&>(rin.transform * vector_cast<vec4>(rp.normal));
    }
};

struct DeviceForceDataSet
{
    sphSphericalGravity *sgravity;
};



struct FluidDataSet
{
    thrust::device_vector<sphGridParam>             params;
    thrust::device_vector<sphStates>                states;
    thrust::device_vector<sphFluidParticle>         particles;
    thrust::device_vector<sphFluidParticleForce>    forces;
    thrust::device_vector<sphHash>                  hashes;
    thrust::device_vector<sphGridData>              grid;

    FluidDataSet()
    {
        params.resize(1);
        states.resize(1);
        particles.resize(SPH_MAX_FLUID_PARTICLES);
        forces.resize(SPH_MAX_FLUID_PARTICLES);
        hashes.resize(SPH_MAX_FLUID_PARTICLES);
        grid.resize(SPH_FLUID_GRID_DIV_3);
    }
    
    void resizeParticles(size_t n)
    {
        particles.resize(n);
        forces.resize(n);
        hashes.resize(n);
    }

    DeviceFluidDataSet getDeviceData()
    {
        DeviceFluidDataSet ddata;
        ddata.params    = params.data().get();
        ddata.states    = states.data().get();
        ddata.particles = particles.data().get();
        ddata.forces    = forces.data().get();
        ddata.hashes    = hashes.data().get();
        ddata.grid      = grid.data().get();
        return ddata;
    }
};


struct RigidDataSet
{
    thrust::device_vector<sphGridParam>         params;
    thrust::device_vector<sphStates>            states;
    thrust::device_vector<sphRigidClass>        classinfo;
    thrust::device_vector<sphRigidInstance>     instances;
    thrust::device_vector<sphRigidParticle>     particles;
    thrust::device_vector<sphRigidUpdateInfo>   updateinfo;
    thrust::device_vector<sphHash>              hashes;
    thrust::device_vector<sphGridData>          grid;

    RigidDataSet()
    {
        params.resize(1);
        states.resize(1);
        classinfo.resize(atomic::CB_END);
        instances.reserve(atomic::ATOMIC_MAX_CHARACTERS);
        particles.reserve(SPH_MAX_RIGID_PARTICLES);
        updateinfo.reserve(SPH_MAX_RIGID_PARTICLES);
        hashes.reserve(SPH_MAX_RIGID_PARTICLES);
        grid.resize(SPH_RIGID_GRID_DIV_3);
    }

    void resizeParticles(size_t n)
    {
        particles.resize(n);
        hashes.resize(n);
    }

    DeviceRigidDataSet getDeviceData()
    {
        DeviceRigidDataSet ddata;
        ddata.params    = params.data().get();
        ddata.states    = states.data().get();
        ddata.classinfo = classinfo.data().get();
        ddata.instances = instances.data().get();
        ddata.particles = particles.data().get();
        ddata.updateinfo= updateinfo.data().get();
        ddata.hashes    = hashes.data().get();
        ddata.grid      = grid.data().get();
        return ddata;
    }
};

struct ForceDataSet
{
    thrust::device_vector<sphSphericalGravity> sgravities;

    ForceDataSet()
    {
        sgravities.reserve(SPH_MAX_SPHERICAL_GRAVITY_NUM);
        sgravities.resize(1);
    }

    DeviceForceDataSet getDeviceData()
    {
        DeviceForceDataSet ddata;
        ddata.sgravity  = sgravities.data().get();
        return ddata;
    }
};

FluidDataSet *h_fluid = NULL;
RigidDataSet *h_rigid = NULL;
ForceDataSet *h_forces = NULL;
sphStates h_states;



__device__ int GetThreadId()
{
    int threadsPerBlock  = blockDim.x * blockDim.y;
    int threadNumInBlock = threadIdx.x + blockDim.x * threadIdx.y;
    int blockNumInGrid   = blockIdx.x  + gridDim.x  * blockIdx.y;
    return blockNumInGrid * threadsPerBlock + threadNumInBlock;
}


__global__ void GClearParticles(DeviceFluidDataSet ps)
{
    const float spacing = 0.009f;
    int i = GetThreadId();
    ps.particles[i].id = i;
    ps.particles[i].alive = 0xffffffff;
    uint w = 128;
    ps.particles[i].position = make_float4(
        spacing*(i%w) - (spacing*w*0.5f),
        spacing*((i/w)%w) + 0.6,
        spacing*(i/(w*w))+0.05f,
        0.0f);
    ps.particles[i].velocity = make_float4(0.0f);

    ps.forces[i].density = 0.0f;
    ps.forces[i].acceleration = make_float4(0.0f);
}

void SPHInitialize()
{
    h_fluid = new FluidDataSet();
    h_rigid = new RigidDataSet();
    h_forces = new ForceDataSet();

    {
        sphParam sph_params;
        sph_params.smooth_len          = 0.02f;
        sph_params.pressure_stiffness  = 200.0f;
        sph_params.rest_density        = 1000.0f;
        sph_params.particle_mass       = 0.001f;
        sph_params.viscosity           = 0.1f;
        sph_params.density_coef        = sph_params.particle_mass * 315.0f / (64.0f * HIP_PI_F * pow(sph_params.smooth_len, 9));
        sph_params.grad_pressure_coef  = sph_params.particle_mass * -45.0f / (HIP_PI_F * pow(sph_params.smooth_len, 6));
        sph_params.lap_viscosity_coef  = sph_params.particle_mass * sph_params.viscosity * 45.0f / (HIP_PI_F * pow(sph_params.smooth_len, 6));
        sph_params.wall_stiffness      = 3000.0f;
        CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL("d_params"), &sph_params, sizeof(sph_params)) );

        sphGridParam grid_params;
        const float grid_len = 5.12f;
        grid_params.grid_dim = make_float4(grid_len, grid_len, sph_params.smooth_len*SPH_FLUID_GRID_DIV_Z, 0.0f);
        grid_params.grid_dim_rcp = make_float4(1.0f) / (grid_params.grid_dim / make_float4(SPH_FLUID_GRID_DIV_X, SPH_FLUID_GRID_DIV_Y, SPH_FLUID_GRID_DIV_Z, 1.0));
        grid_params.grid_pos = make_float4(-grid_len/2.0f, -grid_len/2.0f, 0.0f, 0.0f);
        h_fluid->params[0] = grid_params;

        sphStates stat;
        stat.num_fluid_particles = SPH_MAX_FLUID_PARTICLES;
        stat.num_rigid_particles = 0;
        h_fluid->states[0] = stat;
    }
    {
        sphSphericalGravity h_sg;
        h_sg.position = make_float4(0.0f);
        h_sg.is_active = 1;
        h_sg.inner_radus = 0.5f;
        h_sg.range_radus = 5.12f;
        h_sg.strength = 0.5f;
        h_forces->sgravities[0] = h_sg;
    }

    dim3 dimBlock( SPH_THREAD_BLOCK_X );
    dim3 dimGrid( SPH_MAX_FLUID_PARTICLES / SPH_THREAD_BLOCK_X );
    GClearParticles<<<dimGrid, dimBlock>>>(h_fluid->getDeviceData());
}

void SPHFinalize()
{
    delete h_forces;h_forces=NULL;
    delete h_rigid; h_rigid=NULL;
    delete h_fluid; h_fluid=NULL;
}



struct _UpdateHash
{
    DeviceFluidDataSet dfd;
    _UpdateHash(const DeviceFluidDataSet& v) : dfd(v) {}
    __device__ void operator()(int i) { dfd.updateHash(i); }
};

struct _GridClear
{
    DeviceFluidDataSet dfd;
    _GridClear(const DeviceFluidDataSet& v) : dfd(v) {}
    __device__ void operator()(int i) { dfd.clearGrid(i); }
};

struct _GridUpdate
{
    DeviceFluidDataSet dfd;
    _GridUpdate(const DeviceFluidDataSet& v) : dfd(v) {}
    __device__ void operator()(int i) { dfd.updateGrid(i); }
};

struct _ComputeDensity
{
    DeviceFluidDataSet dfd;
    _ComputeDensity(const DeviceFluidDataSet& v) : dfd(v) {}
    __device__ void operator()(int i) { dfd.computeDensity(i); }
};

struct _ComputeForce
{
    DeviceFluidDataSet dfd;
    _ComputeForce(const DeviceFluidDataSet& v) : dfd(v) {}
    __device__ void operator()(int i) { dfd.computeForce(i); }
};

struct _CountAlives
{
    DeviceFluidDataSet dfd;
    _CountAlives(const DeviceFluidDataSet& v) : dfd(v) {}
    __device__ void operator()(int i) { dfd.countAlives(i); }
};

struct _Integrate
{
    DeviceFluidDataSet dfd;
    DeviceForceDataSet dgd;
    _Integrate(const DeviceFluidDataSet &v, const DeviceForceDataSet &g) : dfd(v), dgd(g) {}

    __device__ void operator()(int i)
    {
        const uint P_ID = i;

        float4 position = dfd.particles[P_ID].position;
        float4 velocity = dfd.particles[P_ID].velocity;
        float4 acceleration = dfd.forces[P_ID].acceleration;
        dfd.hashes[P_ID] = 0;

        //const float3 planes[4] = {
        //    make_float3( 1.0f, 0.0f, 0),
        //    make_float3( 0.0f, 1.0f, 0),
        //    make_float3(-1.0f, 0.0f, 2.56f),
        //    make_float3( 0.0f,-1.0f, 2.56f),
        //};
        //// Apply the forces from the map walls
        //for(uint i = 0 ; i < 4 ; i++)
        //{
        //    float dist = dot(make_float3(position.x, position.y, 1.0f), planes[i]);
        //    acceleration += min(dist, 0.0f) * -d_param.wall_stiffness * make_float4(planes[i].x, planes[i].y, 0.0f, 0.0f);
        //}
        //float4 gravity = make_float4(0.0f, -0.5f, 0.0f, 0.0f);

        acceleration += min(position.z, 0.0f) * -d_params.wall_stiffness * make_float4(0.0f, 0.0f, 0.5f, 0.0f);
        acceleration += make_float4(0.0f, 0.0f, -5.0f, 0.0f);


        // Apply gravity
        for(int i=0; i<SPH_MAX_SPHERICAL_GRAVITY_NUM; ++i) {
            if(!dgd.sgravity[i].is_active) { continue; }

            const float4 center = dgd.sgravity[i].position;
            const float gravity_strength = dgd.sgravity[i].strength;
            const float inner_radius = dgd.sgravity[i].inner_radus;
            const float outer_radius = dgd.sgravity[i].range_radus;

            float4 diff = center-position;
            diff.w = 0.0f;
            float distance = length(diff);
            float4 dir = diff/distance;
            float4 gravity = dir * gravity_strength;

            acceleration += min(distance-inner_radius, 0.0f) * d_params.wall_stiffness * dir;
            acceleration += min(outer_radius-distance, 0.0f) * -d_params.wall_stiffness * dir;
            acceleration += gravity;

            //// kill
            //if(distance-inner_radius < 0.0f) { dfd.hashes[P_ID] = 1; }
        }

        //const float timestep = 1.0f/60.f;
        const float timestep = 0.01f;

        // Integrate
        velocity += timestep * acceleration;
        velocity *= make_float4(0.999);
        if(dot(velocity, velocity) > 1.0f) { velocity *= make_float4(0.98); }
        //velocity.z *= 0.0f;
        position += timestep * velocity;
        //position.z *= 0.0f;

        // Update
        dfd.particles[P_ID].density = dfd.forces[P_ID].density;
        dfd.particles[P_ID].position = position;
        dfd.particles[P_ID].velocity = velocity;
    }
};

void SPHUpdateFluid()
{
    DeviceFluidDataSet dfd = h_fluid->getDeviceData();
    DeviceForceDataSet dgd = h_forces->getDeviceData();
    int num_particles = (int)h_fluid->particles.size();

    thrust::for_each( thrust::make_counting_iterator(0), thrust::make_counting_iterator(num_particles), _UpdateHash(dfd) );
    thrust::sort_by_key(h_fluid->hashes.begin(), h_fluid->hashes.end(), h_fluid->particles.begin());
    thrust::for_each( thrust::make_counting_iterator(0), thrust::make_counting_iterator(SPH_FLUID_GRID_DIV_3), _GridClear(dfd));
    thrust::for_each( thrust::make_counting_iterator(0), thrust::make_counting_iterator(num_particles), _GridUpdate(dfd));

    thrust::for_each( thrust::make_counting_iterator(0), thrust::make_counting_iterator(num_particles), _ComputeDensity(dfd));
    thrust::for_each( thrust::make_counting_iterator(0), thrust::make_counting_iterator(num_particles), _ComputeForce(dfd));
    thrust::for_each( thrust::make_counting_iterator(0), thrust::make_counting_iterator(num_particles), _Integrate(dfd, dgd));
    thrust::sort_by_key(h_fluid->hashes.begin(), h_fluid->hashes.end(), h_fluid->particles.begin());
    thrust::for_each( thrust::make_counting_iterator(0), thrust::make_counting_iterator(num_particles), _CountAlives(dfd));

    const sphStates &stat = h_fluid->states[0];
    h_states.num_fluid_particles = stat.num_fluid_particles;
    if(h_states.num_fluid_particles==1) {
        h_states.num_fluid_particles = h_fluid->hashes[0]==1 ? 1 : 0;
        h_fluid->states[0] = h_states;
    }
    h_fluid->resizeParticles(stat.num_fluid_particles);
}




DeviceBufferObject h_fluid_gl;
DeviceBufferObject h_rigids_gl;
DeviceBufferObject h_light_gl;
thrust::host_vector<sphRigidClass>          h_rigid_class;
thrust::host_vector<sphRigidUpdateInfo>     h_rigid_ui;


void SPHInitializeGLBuffers(int vbo_fluid, int vbo_rigids, int vbo_lightpos)
{
    h_fluid_gl.registerBuffer(vbo_fluid, cudaGraphicsMapFlagsWriteDiscard);
    h_rigids_gl.registerBuffer(vbo_rigids, cudaGraphicsMapFlagsWriteDiscard);
    h_light_gl.registerBuffer(vbo_lightpos, cudaGraphicsMapFlagsWriteDiscard);
}

void SPHFinalizeGLBuffers()
{
    h_light_gl.unregisterBuffer();
    h_rigids_gl.unregisterBuffer();
    h_fluid_gl.unregisterBuffer();
}

void SPHCopyRigidClassInfo(sphRigidClass (&sphcc)[atomic::CB_END])
{
    h_rigid_class.resize(atomic::CB_END);
    thrust::copy(sphcc, sphcc+atomic::CB_END, h_rigid_class.begin());
    h_rigid->classinfo = h_rigid_class;
}


struct _CopyFluid
{
    DeviceFluidDataSet  dfd;
    sphFluidParticle    *gl_partcle;
    float4              *gl_lights;

    _CopyFluid(DeviceFluidDataSet f, sphFluidParticle *glp, float4 *gll)
        : dfd(f), gl_partcle(glp), gl_lights(gll) {}

    __device__ void operator()(int i)
    {
        const uint P_ID = i;
        int pid = dfd.particles[P_ID].id;
        gl_partcle[P_ID] = dfd.particles[P_ID];

        int light_cycle = SPH_MAX_FLUID_PARTICLES/SPH_MAX_LIGHT_NUM;
        if(pid % light_cycle==0) {
            gl_lights[pid/light_cycle] = dfd.particles[P_ID].position;
        }
    }
};

void SPHCopyToGL()
{
    sphFluidParticle *gl_fluid = (sphFluidParticle*)h_fluid_gl.mapBuffer();
    sphRigidParticle *gl_rigid = (sphRigidParticle*)h_rigids_gl.mapBuffer();
    float4 *gl_lights = (float4*)h_light_gl.mapBuffer();

    thrust::for_each( thrust::make_counting_iterator(0), thrust::make_counting_iterator((int)h_fluid->particles.size()),
        _CopyFluid(h_fluid->getDeviceData(), gl_fluid, gl_lights));

    //thrust::copy(d_rigid_p.begin(), d_rigid_p.end(), thrust::device_ptr<sphFluidParticle>(gl_fluid));
    thrust::copy(h_rigid->particles.begin(), h_rigid->particles.end(), thrust::device_ptr<sphRigidParticle>(gl_rigid));

    h_fluid_gl.unmapBuffer();
    h_rigids_gl.unmapBuffer();
    h_light_gl.unmapBuffer();
}



struct _UpdateRigid
{
    DeviceRigidDataSet drd;
    _UpdateRigid(const DeviceRigidDataSet& v) : drd(v) {}
    __device__ void operator()(int i) { drd.updateRigids(i); }
};

void SPHUpdateRigids(const thrust::host_vector<sphRigidInstance> &rigids)
{
    h_rigid->instances = rigids;

    int total = 0;
    for(uint ii=0; ii<rigids.size(); ++ii) {
        int classid = rigids[ii].classid;
        total += h_rigid_class[classid].num_particles;
    }
    h_rigid->particles.resize(total);
    h_states.num_rigid_particles = total;

    int n = 0;
    h_rigid_ui.resize(total);
    for(uint ii=0; ii<rigids.size(); ++ii) {
        int classid = rigids[ii].classid;
        sphRigidClass &cc = h_rigid_class[classid];
        for(uint pi=0; pi<cc.num_particles; ++pi) {
            h_rigid_ui[n+pi].cindex = ii;
            h_rigid_ui[n+pi].pindex = pi;
            h_rigid_ui[n+pi].classid = classid;
            h_rigid_ui[n+pi].owner_handle = rigids[ii].handle;
        }
        n += cc.num_particles;
    }
    h_rigid->updateinfo = h_rigid_ui;

    thrust::for_each(thrust::make_counting_iterator(0), thrust::make_counting_iterator(total), _UpdateRigid(h_rigid->getDeviceData()) );
}


void SPHUpdateGravity(sphSphericalGravity (&sgravity)[ SPH_MAX_SPHERICAL_GRAVITY_NUM ])
{
    thrust::copy(sgravity, sgravity+SPH_MAX_SPHERICAL_GRAVITY_NUM, h_forces->sgravities.begin());
}


void SPHCopyDamageMessageToHost(sphDamageMessage *dst)
{
}



sphStates& SPHGetStates()
{
    return h_states;
}

